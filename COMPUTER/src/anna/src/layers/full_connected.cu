#include "hip/hip_runtime.h"
#include "anna/layers/full_connected.h"
#include "anna/kernels/update_biases.cuh"
#include "anna/cuda/debug.cuh"
#include "anna/cuda/std.cuh"

namespace Anna
{
	namespace Kernel
	{
		__global__ static void cuda_weigh_input(
					const float* input,
					      float* output,
					      float* weights,
					      uint64_t input_count,
					      uint64_t output_count)
		{
			uint16_t idx = threadIdx.x +
			               blockIdx.x * blockDim.x;

			if (idx < output_count)
			{
				const float* p_input     = input;
				      float* p_weights   = weights + idx * input_count;
				const float* p_input_end = input + input_count;

				float sum = 0.0;
				while (p_input != p_input_end)
				{
					sum += *p_input * *p_weights;
					p_input++;
					p_weights++;
				}
				output[idx] += sum;
			}
		}

		__global__ static void cuda_accumulate_gradients(
					      float*   gradients,
					const float*   error,
					const float*   input,
					      uint64_t input_count,
					      uint64_t neurons_count)
		{
			uint16_t  input_idx = threadIdx.x +
			                       blockIdx.x * blockDim.x;
			uint16_t neuron_idx = threadIdx.y +
			                       blockIdx.y * blockDim.y;

			if ( input_idx <  input_count)
			if (neuron_idx < neurons_count)
			{
				float* gradient = gradients  +
				                   input_idx +
				                  neuron_idx * input_count;
				*gradient += error[neuron_idx] * input[input_idx];
			}
		}

		__global__ static void cuda_update_weights(
					      float*   weights,
					const float*   gradients,
					      float    learning_rate,
					      uint64_t input_count,
					      uint64_t neurons_count)
		{
			uint16_t  input_idx = threadIdx.x +
			                       blockIdx.x * blockDim.x;
			uint16_t neuron_idx = threadIdx.y +
			                       blockIdx.y * blockDim.y;

			if ( input_idx <  input_count)
			if (neuron_idx < neurons_count)
			{
				uint64_t idx = input_idx +
				              neuron_idx * input_count;

				weights[idx] += gradients[idx] * learning_rate;
			}
		}

		__global__ static void cuda_calculate_error_back(
						const float* error,
						      float* error_back,
						const float* weights,

						uint64_t input_count,
						uint64_t neurons_count)
		{
			uint16_t idx = threadIdx.x +
			                blockIdx.x * blockDim.x;

			if (idx < input_count)
			{
				const float* p_error  = error;
				const float* p_weight = weights + idx;
				const float* p_error_end = p_error + neurons_count;

				float err = 0.0;
				while (p_error != p_error_end)
				{
					err      += *p_error * *p_weight;
					p_error  += 1;
					p_weight += input_count;
				}

				error_back[idx] = err;
			}
		}
	}

	namespace Layer
	{
		void FullConnected::weigh_input(const Tensor& input)
		{
			uint64_t input_count  =  m_input_shape  .hypervolume();
			uint64_t output_count = m_output.shape().hypervolume();

			#ifdef USE_CUDA
				dim3 block(output_count < 1024 ? output_count : 1024);
				dim3 grid((output_count + block.x - 1) / block.x);

				Kernel::cuda_weigh_input<<<grid, block>>>(
						  input  .data(),
						m_output .data(),
						m_weights.data(),

						input_count,
						output_count);

				cudaCall(hipDeviceSynchronize());
			#else
				// TODO: CPU
			#endif
		}

		void FullConnected::accumulate_gradients(const Tensor& input)
		{
			uint64_t input_count   = m_input_shape.hypervolume();
			uint64_t neurons_count =       m_shape.hypervolume();

			#ifdef USE_CUDA
				dim3 block(  input_count < 32 ?   input_count : 32,
				           neurons_count < 32 ? neurons_count : 32);
				dim3 grid((  input_count + block.x - 1) / block.x,
				          (neurons_count + block.y - 1) / block.y);

				Kernel::cuda_accumulate_gradients<<<grid, block>>>(
						m_gradients.data(),
						m_error    .data(),
						  input    .data(),

						  input_count,
						neurons_count);

				cudaCall(hipDeviceSynchronize());
			#else
				// TODO: CPU
			#endif
		}

		void FullConnected::update_biases(void)
		{
			uint64_t neurons_count = m_shape.hypervolume();

			#ifdef USE_CUDA
				dim3 block(neurons_count < 1024 ? neurons_count : 1024);
				dim3 grid((neurons_count + block.y - 1) / block.y);

				Kernel::cuda_update_biases<<<grid, block>>>(
						m_biases.data(),
						m_error .data(),

						m_hyperparameters->learning_rate(),

						neurons_count);

				cudaCall(hipDeviceSynchronize());
			#else
				// TODO: CPU
			#endif
		}

		void FullConnected::update_weights(void)
		{
			uint64_t input_count   = m_input_shape.hypervolume();
			uint64_t neurons_count =       m_shape.hypervolume();

			#ifdef USE_CUDA
				dim3 block(  input_count < 32 ?   input_count : 32,
				           neurons_count < 32 ? neurons_count : 32);
				dim3 grid((  input_count + block.x - 1) / block.x,
				          (neurons_count + block.y - 1) / block.y);

				Kernel::cuda_update_weights<<<grid, block>>>(
						m_weights  .data(),
						m_gradients.data(),

						m_hyperparameters->learning_rate(),

						input_count,
						neurons_count);

				cudaCall(hipDeviceSynchronize());
			#else
				// TODO: CPU
			#endif
		}

		void FullConnected::calculate_error_back(Tensor& error_back)
		{
			uint64_t   input_count =  m_input_shape  .hypervolume();
			uint64_t neurons_count = m_output.shape().hypervolume();

			dim3 block(input_count < 1024 ? input_count : 1024);
			dim3 grid((input_count + block.x - 1) / block.x);

			#ifdef USE_CUDA
				Kernel::cuda_calculate_error_back<<<grid, block>>>(
						m_error     .data(),
						  error_back.data(),
						m_weights   .data(),

						input_count,
						neurons_count);

				cudaCall(hipDeviceSynchronize());
			#else
				// TODO: CPU
			#endif
		}
	}
}
