#include "hip/hip_runtime.h"
#include "anna/layers/hyperbolic_tangent.h"
#include "anna/cuda/debug.cuh"
#include "anna/cuda/std.cuh"

namespace Anna
{
	namespace Layer
	{
		__global__ static void cuda_activate_kernel(float* data, uint64_t count)
		{
			uint16_t idx = threadIdx.x +
			               blockIdx.x * blockDim.x;

			if (idx < count)
			{
				if      (data[idx] < -20.0) data[idx] = -1;
				else if (data[idx] > +20.0) data[idx] = +1;
				else     data[idx] = tanh(data[idx]);
			}
		}
		void HyperbolicTangent::activate(void)
		{
			uint64_t count = m_shape.hypervolume();

			#ifdef USE_CUDA
				dim3 block(count < 1024 ? count : 1024);
				dim3 grid((count + block.x - 1) / block.x);

				cuda_activate_kernel<<<grid, block>>>(m_output.data(), count);
				cudaCall(hipDeviceSynchronize());
			#else
				// TODO: CPU
			#endif
		}

		__global__ static void cuda_calculate_error_back_kernel(
				const float* error,
				      float* error_back,
				const float* output,
				uint64_t count)
		{
			uint16_t idx = threadIdx.x +
			               blockIdx.x * blockDim.x;

			if (idx < count)
			{
				if      (output[idx] < -20.0) error_back[idx] = 0;
				else if (output[idx] > +20.0) error_back[idx] = 0;
				else
				{
					float cache = tanh(output[idx]);
					error_back[idx] = error[idx] * (1 - cache * cache);
				}
			}
		}
		void HyperbolicTangent::calculate_error_back(Tensor& error_back) const
		{
			uint64_t count = m_shape.hypervolume();

			#ifdef USE_CUDA
				dim3 block(count < 1024 ? count : 1024);
				dim3 grid((count + block.x - 1) / block.x);

				cuda_calculate_error_back_kernel<<<grid, block>>>(
						m_error   .data(),
						error_back.data(),
						m_output  .data(),

						count);
				cudaCall(hipDeviceSynchronize());
			#else
				// TODO: CPU
			#endif
		}
	}
}
