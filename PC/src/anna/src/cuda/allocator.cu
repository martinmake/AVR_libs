#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda/allocator.cuh"

namespace Anna
{
	namespace Cuda
	{
		uint64_t max_allocation_size(void)
		{
			return hipLimitMallocHeapSize;
		}
	}
}
