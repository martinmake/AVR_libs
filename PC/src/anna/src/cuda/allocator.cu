#include <stdio.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda/allocator.cuh"
#include "cuda/debug.cuh"

namespace Anna
{
	namespace Cuda
	{
		void* cuda_malloc(uint64_t size)
		{
			void* d_pointer;

			cudaCall(hipMalloc(&d_pointer, size));

			return d_pointer;
		}

		void cuda_free(void* d_pointer)
		{
			cudaCall(hipFree(d_pointer));
		}

		void cuda_memset(void* d_pointer, uint8_t value, uint64_t size)
		{
			cudaCall(hipMemset(d_pointer, value, size));
		}

		void cuda_memcpy(const void* source_pointer, void* destination_pointer, uint64_t size, CopyDirection direction)
		{
			switch (direction)
			{
				case HOST_TO_DEVICE:   cudaCall(hipMemcpy(destination_pointer, source_pointer, size, hipMemcpyHostToDevice  )); return;
				case DEVICE_TO_HOST:   cudaCall(hipMemcpy(destination_pointer, source_pointer, size, hipMemcpyDeviceToHost  )); return;
				case DEVICE_TO_DEVICE: cudaCall(hipMemcpy(destination_pointer, source_pointer, size, hipMemcpyDeviceToDevice)); return;
			}
		}

		uint64_t cuda_max_allocation_size(void)
		{
			return hipLimitMallocHeapSize;
		}
	}
}
