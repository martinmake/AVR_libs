#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "anna/cuda/device.cuh"
#include "anna/cuda/debug.cuh"

namespace Anna
{
	namespace Cuda
	{
		Device::Device(int initial_idx)
			: m_idx(initial_idx)
		{
			int device_count;

			cudaCall(hipGetDeviceCount(&device_count));
			if (device_count)
				printf("[DEVICE COUNT] %d\n", device_count);
			else
				assert(device_count && "[ERROR] NO DEVICES WERE FOUND");

			hipDeviceProp_t deviceProp;
			cudaCall(hipGetDeviceProperties(&deviceProp, m_idx));
			printf("[DEVICE:%d] NAME:                  %s\n",     m_idx, deviceProp.name);
			printf("[DEVICE:%d] MEMORY CLOCK RATE:     %dKHz\n",  m_idx, deviceProp.memoryClockRate);
			printf("[DEVICE:%d] MEMORY BUS WIDTH:      %d\n",     m_idx, deviceProp.memoryBusWidth);
			printf("[DEVICE:%d] MEMORY PEAK BANDWIDTH: %fGB/s\n", m_idx, deviceProp.memoryClockRate * 2.0 * (deviceProp.memoryBusWidth / 8) / 1.0e6);
			cudaCall(hipSetDevice(m_idx));
		}

		Device::~Device(void)
		{
			cudaCall(hipDeviceReset());
			cudaCall(hipProfilerStop());
		}
	}
}
