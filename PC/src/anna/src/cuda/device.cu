#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "cuda/device.cuh"
#include "cuda/debug.cuh"

namespace Anna
{
	namespace Cuda
	{
		Device::Device(int initial_idx)
			: m_idx(initial_idx)
		{
			hipDeviceProp_t m_deviceProp;
			cudaCall(hipGetDeviceProperties(&m_deviceProp, m_idx));
			printf("[DEVICE:%d] %s\n", m_idx, m_deviceProp.name);
			cudaCall(hipSetDevice(m_idx));
		}

		Device::~Device(void)
		{
		// cudaCall(hipDeviceReset());
		// cudaCall(hipProfilerStop());
		}
	}
}
