#include "hip/hip_runtime.h"
#include "layers/full_connected.h"
#include "cuda/debug.cuh"

extern hipError_t hipConfigureCall(dim3 gridDim, dim3 blockDim, size_t sharedMem=0, hipStream_t stream=0);

namespace Anna
{
	namespace Layer
	{
		__global__
		static void cuda_forward_kernel(
					const float* d_input,
					      float* d_output,
					      float* d_weights,
					      uint64_t input_count,
					      uint64_t output_count);

		void FullConnected::cuda_forward(const Tensor& input)
		{
			uint64_t input_count  = input.shape().hypervolume();
			uint64_t output_count = m_output.shape().hypervolume();

			m_output = m_biases;

			dim3 block(output_count < 1024 ? output_count : 1024);
			dim3 grid((output_count + block.x - 1) / block.x);

			cuda_forward_kernel<<<grid, block>>>(
					input.d_data(),
					m_output.d_data(),
					m_weights.d_data(),
					input_count,
					output_count);

			cudaCall(hipDeviceSynchronize());
		}
		__global__
		static void cuda_forward_kernel(
					const float* d_input,
					      float* d_output,
					      float* d_weights,
					      uint64_t input_count,
					      uint64_t output_count)
		{
			uint16_t idx = threadIdx.x +
			               blockIdx.x * blockDim.x;

			if (idx < output_count)
			{
				const float* d_p_input     = d_input;
				      float* d_p_weights   = d_weights + idx * input_count;
				const float* d_p_input_end = d_input + input_count;

				float sum = 0.0;
				while (d_p_input != d_p_input_end)
				{
					sum += *d_p_input * *d_p_weights;
					d_p_input++;
					d_p_weights++;
				}
				d_output[idx] += sum;
			}
		}
	}
}
