#include "hip/hip_runtime.h"
#include "layers/full_connected.h"
#include "cuda/debug.cuh"
#include "cuda/std.cuh"

namespace Anna
{
	namespace Layer
	{
		__global__ static void cuda_weigh_input_kernel(
					const float* input,
					      float* output,
					      float* weights,
					      uint64_t input_count,
					      uint64_t output_count)
		{
			uint16_t idx = threadIdx.x +
			               blockIdx.x * blockDim.x;

			if (idx < output_count)
			{
				const float* p_input     = input;
				      float* p_weights   = weights + idx * input_count;
				const float* p_input_end = input + input_count;

				float sum = 0.0;
				while (p_input != p_input_end)
				{
					sum += *p_input * *p_weights;
					p_input++;
					p_weights++;
				}
				output[idx] += sum;
			}
		}
		void FullConnected::weigh_input(const Tensor& input)
		{
			uint64_t input_count  =  m_input_shape  .hypervolume();
			uint64_t output_count = m_output.shape().hypervolume();

			#ifdef USE_CUDA
				dim3 block(output_count < 1024 ? output_count : 1024);
				dim3 grid((output_count + block.x - 1) / block.x);

				cuda_weigh_input_kernel<<<grid, block>>>(
						  input  .data(),
						m_output .data(),
						m_weights.data(),

						input_count,
						output_count);

				cudaCall(hipDeviceSynchronize());
			#else
				// TODO: CPU
			#endif
		}

		__global__ static void cuda_accumulate_gradients_kernel(
					      float*   gradients,
					const float*   error,
					const float*   input,
					      uint64_t input_count,
					      uint64_t neurons_count)
		{
			uint16_t  input_idx = threadIdx.x +
			                       blockIdx.x * blockDim.x;
			uint16_t neuron_idx = threadIdx.x +
			                       blockIdx.x * blockDim.x;

			if ( input_idx <  input_count)
			if (neuron_idx < neurons_count)
			{
				float* gradient = gradients  +
				                   input_idx +
				                  neuron_idx * input_count;
				*gradient += error[neuron_idx] + input[input_idx];
			}
		}
		void FullConnected::accumulate_gradients(const Tensor& input)
		{
			uint64_t input_count   = m_input_shape.hypervolume();
			uint64_t neurons_count =       m_shape.hypervolume();

			#ifdef USE_CUDA
				dim3 block(  input_count < 32 ?   input_count : 32,
				           neurons_count < 32 ? neurons_count : 32);
				dim3 grid((  input_count + block.x - 1) / block.x,
				          (neurons_count + block.y - 1) / block.y);

				cuda_accumulate_gradients_kernel<<<grid, block>>>(
						m_gradients.data(),
						m_error    .data(),
						  input    .data(),

						  input_count,
						neurons_count);

				cudaCall(hipDeviceSynchronize());
			#else
				// TODO: CPU
			#endif
		}

		__global__ static void cuda_update_biases_kernel(
					      float* biases,
					const float* error,
					      float  learning_rate,
					      uint64_t neurons_count)
		{
			uint16_t idx = threadIdx.x +
			                blockIdx.x * blockDim.x;

			if (idx < neurons_count)
				biases[idx] += error[idx] * learning_rate;
		}
		void FullConnected::update_biases(void)
		{
			uint64_t neurons_count = m_shape.hypervolume();

			#ifdef USE_CUDA
				dim3 block(neurons_count < 1024 ? neurons_count : 1024);
				dim3 grid((neurons_count + block.y - 1) / block.y);

				cuda_update_biases_kernel<<<grid, block>>>(
						m_biases.data(),
						m_error .data(),

						m_hyperparameters->learning_rate(),

						neurons_count);

				cudaCall(hipDeviceSynchronize());
			#else
				// TODO: CPU
			#endif
		}

		__global__ static void cuda_update_weights_kernel(
					      float*   weights,
					const float*   gradients,
					      float    learning_rate,
					      uint64_t input_count,
					      uint64_t neurons_count)
		{
			uint16_t  input_idx = threadIdx.x +
			                       blockIdx.x * blockDim.x;
			uint16_t neuron_idx = threadIdx.x +
			                       blockIdx.x * blockDim.x;

			if ( input_idx <  input_count)
			if (neuron_idx < neurons_count)
			{
				uint64_t idx = input_idx +
				              neuron_idx * input_count;

				weights[idx] += gradients[idx] * learning_rate;
			}
		}
		void FullConnected::update_weights(void)
		{
			uint64_t input_count   = m_input_shape.hypervolume();
			uint64_t neurons_count =       m_shape.hypervolume();

			#ifdef USE_CUDA
				dim3 block(  input_count < 32 ?   input_count : 32,
				           neurons_count < 32 ? neurons_count : 32);
				dim3 grid((  input_count + block.x - 1) / block.x,
				          (neurons_count + block.y - 1) / block.y);

				cuda_update_weights_kernel<<<grid, block>>>(
						m_weights  .data(),
						m_gradients.data(),

						m_hyperparameters->learning_rate(),

						input_count,
						neurons_count);

				cudaCall(hipDeviceSynchronize());
			#else
				// TODO: CPU
			#endif
		}

		__global__ static void cuda_calculate_error_back_kernel( // TODO
						const float* error,
						      float* error_back,
						const float* weights,

						uint64_t input_count,
						uint64_t neurons_count)
		{
			uint16_t idx = threadIdx.x +
			                blockIdx.x * blockDim.x;

			if (idx < input_count)
			{
				const float* p_error  = error;
				const float* p_weight = weights + idx * input_count;
				const float* p_error_end = p_error + neurons_count;

				float error = 0.0;
				while (p_error != p_error_end)
				{
					error += *p_error * *p_weight;
					p_error  += 1;
					p_weight += input_count;
				}

				error_back[idx] = error;
			}
		}
		void FullConnected::calculate_error_back(Tensor& error_back)
		{
			uint64_t   input_count =  m_input_shape  .hypervolume();
			uint64_t neurons_count = m_output.shape().hypervolume();

			dim3 block(input_count < 1024 ? input_count : 1024);
			dim3 grid((input_count + block.x - 1) / block.x);

			#ifdef USE_CUDA
				cuda_calculate_error_back_kernel<<<grid, block>>>(
						m_error     .data(),
						  error_back.data(),
						m_weights   .data(),

						input_count,
						neurons_count);

				cudaCall(hipDeviceSynchronize());
			#else
				// TODO: CPU
			#endif
		}
	}
}
